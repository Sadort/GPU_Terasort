#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <fstream>

using namespace std;

const unsigned long MASK = 0xFFFF000000000000;

__host__ __device__ bool operator<(const ulong2 &a, const ulong2 &b) {
    if      (a.x < b.x) return true;
    else if (a.x == b.x && (a.y&MASK) <= (b.y&MASK)) return true;
    else return false;
}

typedef struct
{
    ulong2 key;
    unsigned long value;
} mystruct1;

__host__ __device__ bool operator<(const mystruct1 &a, const mystruct1 &b){
    if      (a.key.x < b.key.x) return true;
    else if (a.key.x == b.key.x && a.key.y <= b.key.y) return true;
    else return false;
}

void sort(mystruct1 *H, long int len)
{
    thrust::host_vector<mystruct1> H_vec(H, H+len);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float totalseconds = 0;

    thrust::device_vector<mystruct1> D_vec = H_vec;
    int iterations = 3;
    for(int i = 0; i < iterations; i++)
    {
        hipEventRecord(start, 0);

        thrust::sort(D_vec.begin(), D_vec.end());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totalseconds = totalseconds + milliseconds;
        if (i == iterations - 1) break;
        D_vec = H_vec;
    }
        
    printf("Elapsed time: %f s.\n\n", totalseconds/(iterations*1000));
    H_vec = D_vec;
    for(int i = 0; i < 32; i++)
    {
        cout << H_vec[i].key.x << " ";
    }
    cout << endl;

/*
    thrust::sort(H, H + len);
    for(int i = 0; i < 32; i++)
    {
        cout << H[i] << " ";
    }
    cout << endl;
*/
}

int main(void)
{
    //long int len = 1024*1024*1024;
    //unsigned long *H = (unsigned long *)malloc(len*sizeof(unsigned long));
    //char filename[50] = "ascii_1g_64.out";

/*    const int N = 10;
    long double keys[N] = {1, 4, 4, 5, 4, 5, 3, 1, 4, 5};
    char values[N] = {'1', '1', '2', '1', '3', '2', '1', '2', '4', '3'};
    cout << sizeof(long double) << endl;
    for (int i = 0; i < N; i++)
    {
        cout << keys[i] << "->" << values[i] << endl;
    }
    cout << "After:" << endl;
    thrust::sort_by_key(keys, keys + N, values);
    for (int i = 0; i < N; i++)
    {
        cout << keys[i] << "->" << values[i] << endl;
    }    
*/
    uint64_t len = 512L*1024*1024;
    mystruct1 *H = (mystruct1 *)malloc(len*sizeof(mystruct1));
 
    for (uint64_t i = 0; i < len; i++)
    {
        H[i].key.x = (unsigned long)rand();
        H[i].key.y = (unsigned long)rand();
        H[i].value = (unsigned long)rand();
    } 

    sort(H, len);

/*    thrust::host_vector<unsigned long> h_vec(512*1024*1024);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);
    thrust::device_vector<unsigned long> d_vec = h_vec;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float totalseconds = 0;
    
    for(int i = 0; i < 50; i++)
    {
        hipEventRecord(start, 0);
        thrust::sort(d_vec.begin(), d_vec.end());
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totalseconds = totalseconds + milliseconds;
        d_vec = h_vec;
    }
    printf("Elapsed time: %f s.", totalseconds/50000);
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    for(int i = 0; i < 32; i++)
    {
        cout << h_vec[i] << " ";
    }
    cout << endl;
*/    
    return 0;
}
